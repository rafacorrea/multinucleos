#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define MAX_THREADS 1024
__global__ void multiplicar( float *mat1, float *mat2, float *res, int n) {

  int j=0; int k=0;
  int index = threadIdx.x;
  float maximo = n/(float)blockDim.x;
  for (int i = 0; i < maximo; i++)
  {
      if (index < n)
      {
          for (j = 0; j<n; j++)
          {
              res[index*n+j]=0;

              for (k = 0; k < n; k++)
              {
                  res[index*n+j] += (mat1[index*n + k] * mat2[k*n + j]);
              }
          }
      }
  }
}

void printM(float * data, int rows, int cols)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%.0f ", data[i*cols+j]);
            
        }
        printf("\n");
    }
    printf("\n");
}






int main( int argc, char *argv[] ) {


 if ( argc != 2 ) 

    {

        printf( "usage: %s N (N debe ser < # bloques totales grid size)", argv[0] );
    }
    else
    {
        int n = atoi(argv[1]);


   float *mat1= new float[n*n], *mat2=new float[n*n], *res=new float[n*n];
   float *mat_1, *mat_2, *mat_r;
   float tiempo1, tiempo2;
   hipEvent_t inicio1, fin1, inicio2, fin2; // para medir tiempos como con timestamp



   hipEventCreate(&inicio1); // Se inicializan
   hipEventCreate(&fin1);
   hipEventRecord( inicio1, 0 ); // Se toma el tiempo de inicio


   srand (time(NULL));
        

   // fill the arrays 'a' and 'b' on the CPU
   for (int i=0; i<n*n; i++)
      mat1[i] = mat2[i] = i;


 /* for(int i = 0; i<n*n; i++)
        {
            //mat1[i] = rand()%991 + 10;
		mat1[i] = i;
        }
        
        for(int i = 0; i<n*n; i++)
        {
            //mat2[i] = rand()%991 + 10;
		mat2[i] = i;
            
        }
*/

   // allocate the memory on the GPU
   hipMalloc( (void**)&mat_1, n * n * sizeof(float) );
   hipMalloc( (void**)&mat_2, n * n * sizeof(float) );
   hipMalloc( (void**)&mat_r, n * n * sizeof(float) );

   // copy the arrays 'a' and 'b' to the GPU
   hipMemcpy( mat_1, mat1, n * n  * sizeof(float), hipMemcpyHostToDevice );
   hipMemcpy( mat_2, mat2, n * n  * sizeof(float), hipMemcpyHostToDevice );

   hipEventCreate(&inicio2); // Se inicializan
   hipEventCreate(&fin2);
   hipEventRecord( inicio2, 0 ); // Se toma el tiempo de inicio

   multiplicar<<<1,MAX_THREADS>>>( mat_1, mat_2, mat_r, n );
//   multiplicar<<<1,n>>>( mat_1, mat_2, mat_r, n );

   hipEventRecord( fin2, 0); // Se toma el tiempo final.
   hipEventSynchronize( fin2 ); // Se sincroniza
   hipEventElapsedTime( &tiempo2, inicio2, fin2 );

   // copy the array 'c' back from the GPU to the CPU
   hipMemcpy( res, mat_r, n * n  * sizeof(float), hipMemcpyDeviceToHost );

   // free the memory allocated on the GPU
   hipFree( mat_1 );
   hipFree( mat_2 );
   hipFree( mat_r );

   hipEventRecord( fin1, 0); // Se toma el tiempo final.
   hipEventSynchronize( fin1 ); // Se sincroniza
   hipEventElapsedTime( &tiempo1, inicio1, fin1 );


 	if (res !=0)
        {
	    printf("\nMatriz de Resultado\n\n");
         //   printM(res, n, n);
        }



   free(mat1);
   free(mat2);
   free(res);

   printf("Tiempo cálculo %f ms\n", tiempo2);
   printf("Tiempo total %f ms\n", tiempo1);

   return 0;
  }

        
        //float * res = multiplicar(mat1, mat2, n);


}//Cierre de main
