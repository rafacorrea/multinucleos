
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <omp.h>


int * multiplicar (int * mat1, int *mat2, int n)
{
    
    int * res;
    res = (int*) malloc(n * n * sizeof(int));
    int temp; int i = 0; int j = 0; int k = 0;
//#pragma omp parallel 
    //#pragma omp parallel for schedule(static) private (i,j, k)//collapse(2)
 
 	#pragma omp parallel for ordered

    for (i = 0; i<n; i++)
    {
	#pragma omp ordered
//printf("%d, ", i);
        for (j = 0; j<n; j++)
        {

            temp = 0;
//#pragma omp critical
            for (k = 0; k < n; k++)
            {
                temp += mat1[i*n + k] * mat2[k*n + j];

            }

            res[i*n+j] = temp;

        }

    }
    
    return res;
}

void printM(int * data, int rows, int cols)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%d ", data[i*cols+j]);
            
        }
        printf("\n");
    }
    printf("\n");
}

int main (int argc, char *argv[] )
{
    if ( argc != 2 ) /* argc should be 2 for correct execution */
    {
        /* We print argv[0] assuming it is the program name */
        printf( "usage: %s N", argv[0] );
    }
    else
    {
        int n = atoi(argv[1]);
        int *mat1;
        int *mat2;
        mat1 = (int*) malloc(n * n * sizeof(int));
        mat2 = (int*) malloc(n * n * sizeof(int));
        srand (time(NULL));

   	hipEvent_t inicio, fin;
   	float tiempo;
        

        for(int i = 0; i<n*n; i++)
        {
           // mat1[i] = rand()%991 + 10;
		mat1[i] = i;
        }
        
        for(int i = 0; i<n*n; i++)
        {
           // mat2[i] = rand()%991 + 10;
		mat2[i] = i;
            
        }
        
       /* printM(mat1, n, n);
        printM(mat2, n, n);
	*/

   hipEventCreate( &inicio );
   hipEventCreate( &fin );
   hipEventRecord( inicio, 0 );
        
        int * res = multiplicar(mat1, mat2, n);


   hipEventRecord( fin, 0 );
   hipEventSynchronize( fin );
   hipEventElapsedTime( &tiempo, inicio, fin );


        if (res !=0)
        {
	    printf("\nMatriz de Resultado\n\n");
            printM(res, n, n);
        }
	

 printf("tiempo total en ms: %f\n", tiempo);

        
        return 0;
    }
}


