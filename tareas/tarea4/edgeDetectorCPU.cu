#include "hip/hip_runtime.h"
#include<iostream>
#include<cstdio>
#include<opencv2/core/core.hpp>
#include<opencv2/imgproc/imgproc.hpp>
#include<opencv2/highgui/highgui.hpp>
#include<hip/hip_runtime.h>
#include<math.h>

using namespace std;
using namespace cv;

//int *b


void sumaEulerCPU1(int *a, Mat img, int *c, int N, int M)
{
   // Se calculan valores sin considerar la orilla para3x3.
   for (int i=1; i<N-1; i++)
      for (int j=1; j<M-1; j++)
         c[i*M+j] = ( img.at<uchar>(i*M+j) * a[4] + img.at<uchar>((i-1)*M+j-1) * a[0] + img.at<uchar>((i-1)*M+j) * a[1] + img.at<uchar>((i-1)*M+j+1) * a[2] + img.at<uchar>(i*M+j-1) * a[3] + img.at<uchar>(i*M+j+1) * a[5] + img.at<uchar>((i+1)*M+j-1) * a[6] + img.at<uchar>((i+1)*M+j) * a[7] + img.at<uchar>((i+1)*M+j+1) * a[8] ) / 9;
         // En medio, esq sup izq, arriba, esq sup der, izq, der, esq inf izq, abajo, esq inf der
}


float convolucionCPU1(int *a, Mat img, int *c, int N, int M)
{
   hipEvent_t cpuI, cpuF;
   float cpuT;
   hipEventCreate( &cpuI );
   hipEventCreate( &cpuF );
   hipEventRecord( cpuI, 0 );

   sumaEulerCPU1(a, img, c, N, M);

   hipEventRecord( cpuF, 0 );
   hipEventSynchronize( cpuF );
   hipEventElapsedTime( &cpuT, cpuI, cpuF);
   return cpuT;
}


float combinar(int *a, int *b, Mat &res)
{
    hipEvent_t cpuI, cpuF;
    float cpuT;
    hipEventCreate( &cpuI );
    hipEventCreate( &cpuF );
    hipEventRecord( cpuI, 0 );

    hipEventRecord( cpuF, 0 );
    hipEventSynchronize( cpuF );
    hipEventElapsedTime( &cpuT, cpuI, cpuF);
    int sum;

    /*Se tienen las 2 matrices y se juntan para formar la imagen final en base al algoritmo de PREWITT*/
    for(int y = 1; y < res.rows - 1; y++){
        for(int x = 1; x < res.cols - 1; x++){
	        sum = abs(a[y*res.cols+x])+ abs(b[y*res.cols+x]);
	        sum = sum > 255 ? 255:sum;
            sum = sum < 0 ? 0 : sum;
	        res.at<uchar>(y,x) = sum;
    }
  }
  return cpuT;
}

int main(int argc, char *argv[])
{
	string imagePath;
	
	if(argc < 2)
		imagePath = "test.JPG";
		//imagePath = "space-wallpaper_2880x1800.jpg";
  else
  	imagePath = argv[1];
  	
	//Read input image from the disk in greyscale
	Mat input = imread(imagePath, CV_LOAD_IMAGE_GRAYSCALE);

	if (input.empty())
	{
		cout << "Image Not Found!" << std::endl;
		cin.get();
		return -1;
	}

	//Allow the windows to resize
	namedWindow("Input", WINDOW_NORMAL);

	int  *resX, *resY;
    resX = (int*) malloc(input.rows*input.cols*sizeof(int));
    resY = (int*) malloc(input.rows*input.cols*sizeof(int));
	

	/*Matrices utilizadas para Prewitt*/
	int arregloX[9] = {-1,0,1,-1,0,1,-1,0,1};

	int arregloY[9] = {-1,-1,-1,0,0,0,1,1,1};


	int n=3;


	float tiempo, tiempo2, tiempo3;
	tiempo= convolucionCPU1( arregloX, input , resX, input.rows, input.cols );
    tiempo2= convolucionCPU1( arregloY, input , resY, input.rows, input.cols );

    Mat final;
	
	final = input.clone(); //hagamos un clon
    for(int y = 0; y < input.rows; y++) //recorramos las filas
        for(int x = 0; x < input.cols; x++) //recorramos las columnas
          final.at<uchar>(y,x) = 0.0; //punto inicial


    tiempo3 = combinar(resX, resY, final);

	printf("Tiempo %f: ", tiempo + tiempo2 + tiempo3);

	//Show the input and output
	namedWindow("Input", WINDOW_NORMAL);
	imshow("Input", input);
	namedWindow("Output", WINDOW_NORMAL);
	imshow("Output", final);

	//Wait for key press
	waitKey();

	return 0;
}
