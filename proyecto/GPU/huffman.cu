#include "hip/hip_runtime.h"
#include <stdio.h>   
#include <string.h>  
#include <stdlib.h>  
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h> 
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <cstring>

#include "tree_huff.cuh"

#define FILE_NAME_MAX_LEN 270
#define MAX_FILE_NAME 256
#define N 32
#define M 4

#define THREADS_PER_BLOCK 128
#define WARP_SIZE 32

void add_magic_num(FILE *file);
void add_bit_vector(FILE *file, unsigned char bit_vector[32]);
int  add_size(FILE *file, int freq[MAX_CHARS]);
void add_character_counts(FILE *file, int freq[MAX_CHARS], int num_bytes);


__device__ char * my_strcpy(char *dest, const char *src){
  int i = 0;
  while (src[i] != 0)
  {
    dest[i] = src[i];
    i++;
  }
  return dest;
}

__global__ void encode_byte_stream(char * string, code * code_values, char * res, int * offset, int f_size)
{
	int i = blockIdx.x *blockDim.x + threadIdx.x;
	if(i < f_size)
	{
	    
	    my_strcpy(res + offset[i], code_values[string[i]].path);
	}
	
	
}

__global__ void compressed_bit_stream(char * encoded_byte_stream, unsigned char * res, int f_size)
{
	int i = blockIdx.x *blockDim.x + threadIdx.x;
	if (i < f_size)
	{
	    unsigned char packed = 0;
        for (int count = 0; count < 8; count++) {
           if (encoded_byte_stream[(i*8)+count] == '0') {
              packed <<= 1;
           } else {
              packed = (packed << 1) | 0x01;
           }
         }
         
	     res[i] = packed;
	}
}
int main(int argc, char *argv[]) {

   FILE *file_in, *file_out;
   int freq[MAX_CHARS] = {0}, c = 0, count = 0, num_bytes = 0, ret = 0;
   unsigned char bit_vector[32] = {0x00}, packed = 0;
   char output_file_name[MAX_FILE_NAME] = "", s[MAX_PATH] = "", temp[2*MAX_PATH] = "";
   struct node *tree_head = NULL;
   struct code code_values[MAX_CHARS] = {{-1, {0}, 0}};
   char* texto;
   size_t texto_s;
   long f_size;

   // revisar que haya archivo de entrada
   if (argc != 2) {
      printf("Formato: ./huffman filename\n");
      exit(1);
   }

   // abrir archivo ed entrada
   if ((file_in = fopen(argv[1], "r")) == NULL) {
      printf("No se pudo abrir el archivo.\n");
      exit(1);
   }

   // sacar frecuencia de cada caracter
   while ((c = fgetc(file_in)) != EOF) {
      freq[c]++;
   }

   // cerrar archivo de entrada
   fclose(file_in);

   for (count = 0; count < MAX_CHARS; count++) {
      if (freq[count] > 0) {
         bit_vector[count / 8] |= (1 << (count % 8));
      }
   }


   if ((strlen(argv[1])+strlen(".huff")) >= MAX_FILE_NAME) {
      printf("Input file name too long.  Output file cannot be generated.\n");
      exit(1);
   }

   // crear archivo de salida
   strncpy(output_file_name, argv[1], MAX_FILE_NAME);
   strncat(output_file_name, ".huff", MAX_FILE_NAME);

   // abrir archivo de salida
   if ((file_out = fopen(output_file_name, "w")) == NULL) {
      printf("Output file failed to open.\n");
      exit(1);
   }


   add_magic_num(file_out);
   add_bit_vector(file_out, bit_vector);
   num_bytes = add_size(file_out, freq);
   add_character_counts(file_out, freq, num_bytes);

   // construir arbol
   tree_head = generate_tree(freq);

   // generar huffman codes
   build_codes(tree_head, code_values, s, 0);

   // abrir arhchivo de entrada
   if ((file_in = fopen(argv[1], "r")) == NULL) {
      printf("Failed to open the input file.\n");
      exit(1);
   }


   //NUEVO **************************************
   
   hipEvent_t cpuI, cpuF;
   float cpuT;
   
   hipEventCreate( &cpuI );
    hipEventCreate( &cpuF );
    hipEventRecord( cpuI, 0 );
     
   fseek(file_in, 0, SEEK_END);
   f_size = ftell(file_in);
   fseek(file_in, 0, SEEK_SET);
   
   char *string = (char *)malloc(sizeof(char) * f_size); 
   int * offset = (int *)malloc(sizeof(int) * f_size);

   int i = 0;

   while ((c = fgetc(file_in)) != EOF) {
      offset[i] = code_values[c].len;
      string[i] = c;
      i++;
      }
   char *d_string;
   
   hipMalloc<char>(&d_string, sizeof(char) * f_size);
   hipMemcpy(d_string, string, f_size*sizeof(char), hipMemcpyHostToDevice );
   
   thrust::device_vector<int> d_offset(offset, offset+f_size);   
  	
   thrust::exclusive_scan(d_offset.begin(), d_offset.end(), d_offset.begin()); // in-place scan
    
   thrust::copy(d_offset.begin(), d_offset.end(), offset);   
   int last = offset[f_size - 1] + code_values[string[f_size-1]].len;
   //padding
   if (last%8 != 0)
   {
      last += 8-(last%8);
   }

   int * d_offset2 = thrust::raw_pointer_cast( d_offset.data());
   
   
   char * d_encoded_byte_stream;
   char * encoded_byte_stream = (char *)malloc(sizeof(char) * last);
   hipMalloc<char>(&d_encoded_byte_stream, sizeof(char)*last);
   hipMemset(d_encoded_byte_stream, '0', sizeof(char)*last);
   code * d_code_values;
   hipMalloc<code>(&d_code_values, sizeof(code)*MAX_CHARS);   
   hipMemcpy(d_code_values, code_values, MAX_CHARS*sizeof(code), hipMemcpyHostToDevice );   
   
   
   //thrust::device_delete(d_offset2);
   int blocks;
   blocks = ceil((float)f_size/THREADS_PER_BLOCK);
   
   encode_byte_stream<<<blocks,THREADS_PER_BLOCK>>>(d_string, d_code_values, d_encoded_byte_stream, d_offset2, f_size);
   hipFree(d_string);
   hipFree(d_code_values);
   //d_offset.clear();
  // d_offset.shrink_to_fit();
   
   //hipFree(d_offset2);
   
   hipMemcpy(encoded_byte_stream, d_encoded_byte_stream, last*sizeof(char), hipMemcpyDeviceToHost);
   
   int finalSize = last/8;
  
   unsigned char * d_encoded_bit_stream;
   unsigned char * encoded_bit_stream = (unsigned char *)malloc(finalSize * sizeof(unsigned char));
   
   hipMalloc<unsigned char>(&d_encoded_bit_stream, sizeof(unsigned char)*finalSize);
   blocks = ceil((float)finalSize/THREADS_PER_BLOCK);
   compressed_bit_stream<<<blocks, THREADS_PER_BLOCK>>>(d_encoded_byte_stream, d_encoded_bit_stream, finalSize);
   hipMemcpy(encoded_bit_stream, d_encoded_bit_stream, finalSize*sizeof(unsigned char), hipMemcpyDeviceToHost);
   
   
    
   fwrite(encoded_bit_stream, sizeof(unsigned char), finalSize, file_out);
   
   hipEventRecord( cpuF, 0 );
   hipEventSynchronize( cpuF );
   hipEventElapsedTime( &cpuT, cpuI, cpuF);
    
   printf("Tiempo %f: ", cpuT);
   hipFree(d_encoded_byte_stream);
   hipFree(d_encoded_bit_stream);
   
   free(encoded_bit_stream);
   free(encoded_byte_stream);
   free(string);
   free(offset);
   /*
   fseek(file_in, 0, SEEK_SET);
   
   //TERMINA NUEVO ************************************
   
   
   
   // basado en el codigo huffman llenar archivo
   while ((c = fgetc(file_in)) != EOF) {
      strncat(temp, code_values[c].path, 2*MAX_PATH);
      
      if (strlen(temp) < 8) {
         continue;
      }

      // escribir a la salida
      while (strlen(temp) > 8) {
         for (count = 0; count < 8; count++) {
            if (temp[count] == '0') {
               packed <<= 1;
            } else {
               packed = (packed << 1) | 0x01;
            }
         }

         fwrite(&packed, sizeof(unsigned char), 1, file_out);
         packed = 0;
         strcpy(temp, &temp[8]);
      }
   }

   packed = 0;


   if (strlen(temp) > 0) {
      
      for (count = 0; count < strlen(temp); count++) {
         if (temp[count] == '0') {
            packed <<= 1;
         } else {
            packed = (packed << 1) | 0x01;
         }
      }

      // padding
      for (count = strlen(temp); count < 8; count++) {
         packed = (packed << 1) ;
      }

      // escribir al archivo
      fwrite(&packed, sizeof(unsigned char), 1, file_out);
   }
   */
   // cerrar archivo entrada
   if ((ret = fclose(file_in)) != 0) {
      printf("Failed to close the input file.");
   }

   //cerrar archivo salida
   if ((ret = fclose(file_out)) != 0) {
      printf("Failed to close the output file.");
   }

   free_tree(tree_head);

   return 0;
}

void add_magic_num(FILE *file) {

   unsigned char magic_num[4] = {0x4C,0x70,0xF0,0x7C};
   int i = 0, ret = 0;

   for (i = 0; i < 4; i++) {
      if ((ret = fprintf(file, "%c", magic_num[i])) != 1) {
         printf("Failure to add magic number to output file.\n");
         exit(1);
      }
   }

   return;
}

void add_bit_vector(FILE *file, unsigned char bit_vector[32]) {
   unsigned char c = 0;
   int i = 0, ret = 0;


   for (i = 0; i < 32; i++) {

      c = 0x00;
      c |= ((bit_vector[i] & 0x01) << 7);
      c |= ((bit_vector[i] & 0x02) << 5);
      c |= ((bit_vector[i] & 0x04) << 3);
      c |= ((bit_vector[i] & 0x08) << 1);
      c |= ((bit_vector[i] & 0x10) >> 1);
      c |= ((bit_vector[i] & 0x20) >> 3);
      c |= ((bit_vector[i] & 0x40) >> 5);
      c |= ((bit_vector[i] & 0x80) >> 7);

      if ((ret = fprintf(file,"%c", c)) != 1) {
         printf("Failure to output bit vector number.\n");
         exit(1);
      }
   }

   return;
}

int add_size(FILE *file, int freq[MAX_CHARS]) {

   int i = 0, num_bytes = 0, ret = 0;


   for (i = 0; i < MAX_CHARS; i++) {
      if (freq[i] & 0xFF000000) {
         num_bytes = 4;
      } else if ((freq[i] & 0x00FF0000) && num_bytes < 4) {
         num_bytes = 3;
      } else if ((freq[i] & 0x0000FF00) && num_bytes < 3) {
         num_bytes = 2;
      } else if ((freq[i] & 0x000000FF) && num_bytes < 2) {
         num_bytes = 1;
      }
   }


   if ((ret = fprintf(file, "%c", num_bytes)) != 1) {
      printf("Failure to output size of the frequency byte number.\n");
      exit(1);
   }

   return num_bytes;
}

void add_character_counts(FILE *file, int freq[MAX_CHARS], int num_bytes) {

   char *ptr = 0;
   int i = 0, ret = 0, j = 0;


   for (i = 0; i < MAX_CHARS; i++) {

      if (freq[i] == 0) {
         continue;
      }

      ptr = (char *)&freq[i];


      for (j = (num_bytes - 1); j >= 0 ; j--) {

         if ((ret = fprintf(file, "%c", ptr[j])) != 1) {
            printf("Failure to output the freqency byte.\n");
            exit(1);
         }
      }
   }

   return;
}

